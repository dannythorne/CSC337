

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void kernel( int* b, int* t)
{
  *b = gridDim.x; // Blocks in the grid
  *t = blockDim.x; // Treads per block
}

int main()
{
  int b;
  int* d_b;
  int t;
  int* d_t;

  // store in d_b the address of a memory
  // location on the device
  hipMalloc( (void**)&d_b, sizeof(int));
  hipMalloc( (void**)&d_t, sizeof(int));

  kernel<<<1,1>>>(d_b,d_t);

  hipMemcpy( &b, d_b, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy( &t, d_t, sizeof(int), hipMemcpyDeviceToHost);

  hipFree(d_b);
  hipFree(d_t);

  cout << "Num blocks           : " << b << endl;
  cout << "Num threads per block: " << t << endl;

  return 0;
}
