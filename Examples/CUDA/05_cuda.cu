

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void kernel( int* b, int* t)
{
  if( !threadIdx.x)
  {
    *b = blockDim.x; // num threads per block
  }
  t[threadIdx.x] = threadIdx.x;
}

int main()
{
  int numthreads = 4;

  int b;
  int* t;

  t = new int[numthreads];

  int* d_b; // pointer to device memory
  int* d_t; // pointer to device memory

  hipMalloc( (void**)&d_b, sizeof(int));
  hipMalloc( (void**)&d_t, numthreads*sizeof(int));

  kernel<<<1,numthreads>>>( d_b, d_t);

  hipMemcpy( &b, d_b, sizeof(int)
            , hipMemcpyDeviceToHost);

  hipMemcpy( t, d_t, numthreads*sizeof(int)
            , hipMemcpyDeviceToHost);

  cout << "blockDim.x = " << b << endl;

  int thread;
  for( thread=0; thread<numthreads; thread++)
  {
    cout << "thread " << thread
         << ": " << t[thread]
         << endl;
  }

  hipFree(d_t);
  hipFree(d_b);

  return 0;
}
